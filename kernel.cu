#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

inline float getRandomFloat() {
    return (float)rand() / (float)RAND_MAX;
}

void initMatrix(float* a, int rows, int cols) {
    for (int i = 0; i < rows; ++i)
        for (int j = 0; j < cols; ++j)
            a[i * cols + j] = /*getRandomFloat()*/ i;
}

void matrixMultiply(float* c, float* a, float* b, const int aRows, const int aCols, const int bRows, const int bCols) {
    assert(aCols == bRows && "invalid dimensions for matrix multiplication!");
    const int cRows = aRows;
    const int cCols = bCols;
    for (int i = 0; i < cRows; ++i) {
        for (int j = 0; j < cCols; ++j) {
            int cIdx = i * cCols + j;
            c[cIdx] = 0.0;
            for (int k = 0; k < aCols; ++k) {
                int aIdx = i * aCols + k;
                int bIdx = k * bCols + j;
                c[cIdx] += a[aIdx] * b[bIdx];
            }
        }
    }
}

__global__
void matrixMultiplyGPU(float* c, float* a, float* b, const int aRows, const int aCols, const int bRows, const int bCols) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    //printf("compute index: (%d, %d)...\n", i, j);
    const int cRows = aRows;
    const int cCols = bCols;
    int idx = i * cCols + j;
    if (idx < cRows * cCols) {
        c[idx] = 0.0;
        for (int k = 0; k < aCols; ++k) {
            int aIdx = i * aCols + k;
            int bIdx = k * bCols + j;
            c[idx] += a[aIdx] * b[bIdx];
        }
    }
}

void printMatrix(float* mat, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%f ", mat[i*cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(){
    hipSetDevice(0);
    srand((float)time(NULL));

    // set dimensions
    const int aRows = 4;
    const int aCols = 3;
    const int bRows = 3;
    const int bCols = 4;
    const int cRows = 4;
    const int cCols = 4;
    
    // allocate space for matrices
    float* a = (float*)malloc(aRows * aCols * sizeof(float));
    float* b = (float*)malloc(bRows * bCols * sizeof(float));
    float* c = (float*)malloc(cRows * cCols * sizeof(float));

    // initialize matrices
    initMatrix(a, aRows, aCols);
    initMatrix(b, bRows, bCols);

    // allocate global memory
    float* device_a;
    float* device_b;
    float* device_c;
    hipMalloc((void**)&device_a, aRows * aCols * sizeof(float));
    hipMalloc((void**)&device_b, bRows * bCols * sizeof(float));
    hipMalloc((void**)&device_c, cRows * cCols * sizeof(float));

    // copy over data from host to device
    hipMemcpy(device_a, a, aRows * aCols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, bRows * bCols * sizeof(float), hipMemcpyHostToDevice);

    // execute matrix multiplication on device
    //matrixMultiply(c, a, b, aRows, aCols, bRows, bCols);
    dim3 numThreads(cRows, cCols);
    matrixMultiplyGPU << <1, numThreads>> > (device_c, device_a, device_b, aRows, aCols, bRows, bCols);
    hipDeviceSynchronize();

    // copy over data from host to device
    hipMemcpy(c, device_c, cRows * cCols * sizeof(float), hipMemcpyDeviceToHost);

    printMatrix(a, aRows, aCols);
    printMatrix(b, bRows, bCols);
    printMatrix(c, cRows, cCols);

    free(a);
    free(b);
    free(c);
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
    return 0;
}
