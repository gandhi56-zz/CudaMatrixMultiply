
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

inline float getRandomFloat() {
    return (float)rand() / (float)RAND_MAX;
}

void initMatrix(float* a, int rows, int cols) {
    for (int i = 0; i < rows; ++i)
        for (int j = 0; j < cols; ++j)
            a[i * cols + j] = /*getRandomFloat()*/ i;
}

void matrixMultiply(float* c, float* a, float* b, const int aRows, const int aCols, const int bRows, const int bCols) {
    assert(aCols == bRows && "invalid dimensions for matrix multiplication!");
    const int cRows = aRows;
    const int cCols = bCols;
    for (int i = 0; i < cRows; ++i) {
        for (int j = 0; j < cCols; ++j) {
            int cIdx = i * cCols + j;
            c[cIdx] = 0.0;
            for (int k = 0; k < aCols; ++k) {
                int aIdx = i * aCols + k;
                int bIdx = k * bCols + j;
                c[cIdx] += a[aIdx] * b[bIdx];
            }
        }
    }
}

void printMatrix(float* mat, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%f ", mat[i*cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(){
    srand((float)time(NULL));

    const int aRows = 4;
    const int aCols = 3;
    const int bRows = 3;
    const int bCols = 4;
    const int cRows = 4;
    const int cCols = 4;
    float* a = (float*)malloc(aRows * aCols * sizeof(float));
    float* b = (float*)malloc(bRows * bCols * sizeof(float));
    float* c = (float*)malloc(cRows * cCols * sizeof(float));

    initMatrix(a, aRows, aCols);
    initMatrix(b, bRows, bCols);
    matrixMultiply(c, a, b, aRows, aCols, bRows, bCols);

    printMatrix(a, aRows, aCols);
    printMatrix(b, bRows, bCols);
    printMatrix(c, cRows, cCols);

    free(a);
    free(b);
    free(c);
    return 0;
}
